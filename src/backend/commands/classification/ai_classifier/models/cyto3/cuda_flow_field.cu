
#include <hip/hip_runtime.h>
///
/// \file      cuda_flow_field.cu
/// \author    Joachim Danmayr
/// \date      2025-08-03
///
/// \copyright Copyright 2019 Joachim Danmayr
///            This software is licensed for **non-commercial** use only.
///            Educational, research, and personal use are permitted.
///            For **Commercial** please contact the copyright owner.
///

#ifdef WITH_CUDA

#include <cuda_runtime.h>                // Core CUDA runtime API (for cudaMalloc, cudaMemcpy, etc.)
#include <device_launch_parameters.h>    // For thread/block indexing macros
#include <math_constants.h>              // Optional, for constants like CUDART_INF_F, if needed
#include <cmath>                         // For std::fabs, std::floor if you use host code (but inside __device__ use CUDA math functions)
#include <cstdio>

///
/// \brief
/// \author     Joachim Danmayr
/// \param[in]
/// \param[out]
/// \return
///
__device__ float bilinearInterpolate(const float *data, int width, int height, float x, float y)
{
  int x0 = floorf(x);
  int x1 = min(x0 + 1, width - 1);
  int y0 = floorf(y);
  int y1 = min(y0 + 1, height - 1);

  float dx = x - x0;
  float dy = y - y0;

  float val00 = data[y0 * width + x0];
  float val10 = data[y0 * width + x1];
  float val01 = data[y1 * width + x0];
  float val11 = data[y1 * width + x1];

  float val0 = val00 * (1 - dx) + val10 * dx;
  float val1 = val01 * (1 - dx) + val11 * dx;

  return val0 * (1 - dy) + val1 * dy;
}

///
/// \brief
/// \author     Joachim Danmayr
/// \param[in]
/// \param[out]
/// \return
///
__global__ void followFlow(const float *flowX, const float *flowY, const float *mask, int width, int height, float stepSize, int numSteps,
                           float epsilon, float *outputX, float *outputY, float maskThreshold)
{
  int px = blockIdx.x * blockDim.x + threadIdx.x;
  int py = blockIdx.y * blockDim.y + threadIdx.y;

  if(mask[py * width + px] < maskThreshold) {
    return;
  }

  if(px >= width || py >= height) {
    return;
  }

  float x = static_cast<float>(px);
  float y = static_cast<float>(py);

  for(int i = 0; i < numSteps; ++i) {
    float fx = bilinearInterpolate(flowX, width, height, x, y);
    float fy = bilinearInterpolate(flowY, width, height, x, y);

    if(fabsf(fx) < epsilon && fabsf(fy) < epsilon) {
      break;
    }

    x += stepSize * fx;
    y += stepSize * fy;

    x = fminf(fmaxf(x, 0.0f), static_cast<float>(width - 1));
    y = fminf(fmaxf(y, 0.0f), static_cast<float>(height - 1));
  }

  outputX[py * width + px] = x;
  outputY[py * width + px] = y;
}

///
/// \brief
/// \author     Joachim Danmayr
/// \param[in]
/// \param[out]
/// \return
///
extern "C" void cudaFlowIterationKernel(const float *flowX, const float *flowY, const float *mask, int width, int height, float stepSize,
                                        int numSteps, float epsilon, float *outputX, float *outputY, float maskThreshold)
{
  dim3 block(2, 2);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

  followFlow<<<grid, block>>>(flowX, flowY, mask, width, height, stepSize, numSteps, epsilon, outputX, outputY, maskThreshold);

  cudaDeviceSynchronize();    // optional but good for debugging
}
#endif
